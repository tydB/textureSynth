#include "hip/hip_runtime.h"
#include <windows.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
//#include <GL/glut.h>
#include "glut.h" // I have copied glut.h into the current directory
#include <hip/hip_runtime.h>

#define WINSIZE 512 // Size of the window
#define YES 1
#define NO 0

/* Define a colour in terms of the r, g, b components*/
typedef struct {
	unsigned char red;
	unsigned char green;
	unsigned char blue; 
} rgb;

/* Create some space to render the image into */
rgb image[WINSIZE*WINSIZE];
rgb mask[WINSIZE*WINSIZE];
rgb* sample;
int sampleWidth;
int sampleHeight;

// Kernel that executes on the CUDA device  
__global__ void render(rgb* image, int width, int height) {
	int x = threadIdx.x;
	int y = blockIdx.x;
	image[y*width+x].red = x % width;
	image[y*width+x].green = y % width;
	image[y*width+x].blue = 0;
}
void loadBMP() {
	printf("\nLoading Image\n");
	unsigned char header[54];
	unsigned int dataStart;
	unsigned int width;
	unsigned int height;
	int imageSize;
	// pixel data
	// unsigned char *data;
	FILE * file = fopen("smallRocks.bmp","rb");
	if (!file) {
		printf("Texture could not be found\n");
		return;
	}
	if (fread(header, 1, 54, file) != 54) { // If not 54 bytes read : problem
		printf("Error with the texture file\n");
		return;
	}
	// get header data
	dataStart = *(int*)&(header[10]);
	imageSize = *(int*)&(header[34]);
	width = *(int*)&(header[18]);
	height = *(int*)&(header[22]);
	int padding = (width*3)%4;

	// get memory for the sampleImage
	sampleWidth = width;
	sampleHeight = height;
	sample = (rgb*)malloc(sizeof(rgb) * sampleWidth * sampleHeight);

	printf("dataStart %d\n", dataStart);
	printf("imageSize %d\n", imageSize);
	printf("width     %d\n", width);
	printf("height    %d\n", height);
	printf("padding   %d\n", padding);

	//get to pixel data

	// create array for pixel data
	unsigned char* data = (unsigned char*)malloc(width * height * 3 * sizeof(unsigned char));
	// get to the pixel data
	fread(data, 1, dataStart - 54, file); // do this better---------------------------------------------------------------------
	// Read the pixel data
	fread(data,1,width * height * 3,file);
	fclose(file);

	// // move the data from the buffer to the pattern
	for (int i = 0; i < width; ++i) {
		for (int j = 0; j < height; ++j) {
			sample[i * width + j].red = (unsigned char)data[(i * width + j) * 3   + 2];
			sample[i * width + j].green = (unsigned char)data[(i * width + j) * 3 + 1];
			sample[i * width + j].blue = (unsigned char)data[(i * width + j) * 3  + 0];
		}
	}
	// // free the buffer
	free(data);
}
void setPixel(int x, int y, rgb color) {
	image[y * WINSIZE + x].red = color.red;
	image[y * WINSIZE + x].green = color.green;
	image[y * WINSIZE + x].blue = color.blue;
	mask[y * WINSIZE + x].red = 255;
	mask[y * WINSIZE + x].green = 255;
	mask[y * WINSIZE + x].blue = 255;
}
rgb getSampleColor(int x, int y) {
	return sample[y * sampleHeight + x];
}
void seed(int seedSize) {
	int x = 60;
	int y = 60;
	for (int sampleI = 0; sampleI < seedSize; sampleI++) {
		for (int sampleJ = 0; sampleJ < seedSize; sampleJ++) {
			setPixel(WINSIZE / 2 + sampleI - 1,
				WINSIZE / 2 + sampleJ - 1,
				getSampleColor(x + sampleI,y + sampleJ));
		}
	}
}
vector<rgb> getNextPixels() {
	return NULL;
}
void synthTexture(int nSize) {

}
//***** OpenGL code
// also contains some CUDA code 
void showimage(void) {
	// This code allocates memory on the graphics card.
	// The amount it allocates is WINSIZE*WINSIZE*sizeof(rgb), which is enough space for every pixel to display
	// rgb * image_Device;
	// hipMalloc((void **) &image_Device, WINSIZE*WINSIZE*sizeof(rgb));
	// // The number of threads to pass to CUDA will be one for each pixel.
	// int block_size = WINSIZE;  
	// int n_blocks = WINSIZE;
	// // This calls CUDA to run the render function on the graphics card.
	// // CUDA should start n_blocks times block_size threads and pass in the appropriate ID to each thread
	// // Note that the order the threads execute is not known.  A pixel later in the image may be rendered before
	// // a pixel earlier in the image
	// render <<< n_blocks, block_size >>> (image_Device, WINSIZE, WINSIZE);
	// // This copies the generated image from the graphics card back to the CPU RAM where it can be drawn
	// // (somewhat ironic since it is going to be put into OpenGL which sends it back to the graphics card)
	// hipMemcpy(image, image_Device, WINSIZE*WINSIZE*sizeof(rgb), hipMemcpyDeviceToHost);
 // 	// Free the memory that has been allocated
	// hipFree(image_Device);
	// This is OpenGL code that makes a display list that draws the rendered image.
	glNewList(1,GL_COMPILE_AND_EXECUTE);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glDrawPixels(WINSIZE,WINSIZE, GL_RGB, GL_UNSIGNED_BYTE, image);
	glDrawPixels(sampleWidth,sampleHeight, GL_RGB, GL_UNSIGNED_BYTE, sample);
	glEndList();
}
// Display call back function clears the screen and draws the rendered image (which at this point is just a picture)
void display(void) {
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);  
	glCallList(1);
	glutSwapBuffers();
}
// This makes the program exit when the user presses 'q'
void keyboard(unsigned char key, int x, int y) {
	switch (key) {
		case 'q': /* user selected quit */
			free(sample);
			exit(0);
			break;
		}
}
// This function animates the four spheres.  It basically
// updates the positions assumming a constant motion and checks for
// intersections with the edges of the world.
// All of this code runs on the CPU (rather than the GPU)
void idle(void) {
	showimage(); // render the image on the graphics card
	glutPostRedisplay(); // request OpenGL show the newly rendered image
}
// main routine that executes on the host  
int main(void) {
	// First print out a little information about the device
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	printf("There are %d devices\n", deviceCount);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0); // get the properties from the first devie
	printf("Name: %s\n", deviceProp.name);
	printf("Total Global Memory: %d\n", deviceProp.totalGlobalMem);
	printf("Shared memory per block: %d\n", deviceProp.sharedMemPerBlock);
	printf("Constant Memory: %d\n", deviceProp.totalConstMem);
	printf("CUDA %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("Clock Rate: %d kHz\n", deviceProp.clockRate);
	printf("Warp size: %d\n", deviceProp.warpSize);

	// This code folowing here is OpenGL setup code
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutCreateWindow("Texture Synthesis");
	glutReshapeWindow(WINSIZE,WINSIZE);

	// get sample image
	loadBMP();
	// seed
	seed(20);
	// synthesis
	synthTexture(5);
	//These callback functions tell OpenGL what to call when there is something to do
	glutDisplayFunc(display); // set the display call back function
	glutKeyboardFunc(keyboard); // set the keyboard call back function
	// glutIdleFunc(idle);  // set the idle call back function
	srand( (unsigned)time( NULL ) );
	showimage(); // Draw the first time
	glutMainLoop(); // pass control over to OpenGL which will call the appropriate call back function when needed
}